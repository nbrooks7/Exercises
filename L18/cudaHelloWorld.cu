#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

// This is my DEVICE function
// __global__ mean this function is visible to the host
__global__ void kernelHelloWorld(){

	int thread = threadIdx.x; //local thread number in a block 
	int block = blockIdx.x; //block number
	
	printf("Hello World form thread %d of block %d!\n", thread, block);
}


int main(int argc, char** argv){
	
	int Nblocks = 10; // number of blocks
	int Nthreads = 3; // number of threads per block


	// run the function 'kernelHElloWorld' on the DEVICE
	kernelHelloWorld <<< Nblocks, Nthreads >>> ();

	//wait for the DEVICE function to complete before moving on
	hipDeviceSynchronize();

	return 0;

}
