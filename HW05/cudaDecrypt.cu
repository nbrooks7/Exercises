#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"


int main (int argc, char **argv) {
/* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
   __device__ unsigned int modprodDev(unsigned int a, unsigned int b, unsigned int p) {
        unsigned int za = a;
        unsigned int ab = 0;

        while (b > 0) {
             if (b%2 == 1) ab = (ab +  za) % p;
                za = (2 * za) % p;
                b /= 2;
        }
        return ab;
    }
  
   __device__ unsigned int modExpDev(unsigned int a, unsigned int b, unsigned int p) {
         unsigned int z = a;
         unsigned int aExpb = 1;

        while (b > 0) {
             if (b%2 == 1) aExpb = modprodDev(aExpb, z, p);
                 z = modprodDev(z, z, p);
                 b /= 2;
        }
        return aExpb;
    }

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  FILE* file = fopen("public_key.txt","r");
  unsigned int *pc = (int *) malloc(4*sizeof(unsigned int));

  for (int m = 0;m<4;m++){
      fscanf(file, "%d", pc+m);
  }
  
  n = pc[0];
  p = pc[1];
  g = pc[2];
  h = pc[3];

  fclose(file);
  free(pc);

  FILE* file2 = fopen("message.txt", "r");
  fscanf(file2, "%d", &Nints);

  unsigned int *Zmessage = (unsigned int *) malloc(Nints*sizeof(unsigned int));
  unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));

  for (int n = 0; n<Nints; n++){
     fscanf(file2, "%d %d", &Zmessage[n],&a[n] );
  }
  fclose(file2);
  
  
  // find the secret key
  if (x==0 || modExpDev(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (modExpDev(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        x=i+1;
      } 
    }
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }

  /* Q3 After finding the secret key, decrypt the message */
  int bufferSize = 1024;
  unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
  unsigned int Nchars = ((n-1)/8)*Nints; 
  ElGamalDecrypt(Zmessage,a,Nints,p,x);
  convertZToString(Zmessage, Nints, message, Nchars);
  printf("Decrypted Message = \"%s\"\n", message);
  
  //for (int n =0;n<Nints;n++){
      //printf("message[%d] = %d\n",n,message[n]);
  //}
  //for (int m =0;m<Nints;m++){
      //printf("a[%d] = %d\n",m,a[m]);
  //}

  free(Zmessage);
  free(a);
  free(message);
  return 0;
}
